
#include <hip/hip_runtime.h>
extern "C"
__global__ void JCudaTextureKernel(
        float * output,
        hipTextureObject_t texObj,
        int width,
        int height,
        float theta
    ) {
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    // Transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    // Read from texture and write to global memory
    output[y * width + x] = tex2D<float>(texObj, tu, tv);
}